#include "hip/hip_runtime.h"
#include "BoidScene.h"
#include "Common.h"
#include <iostream>

#if CUDA
#define THREADS_PER_BLOCK 256
#endif
//k value
#define MAX_DISTANCE 65.0f
#define MAX_DISTANCE_SQR MAX_DISTANCE * MAX_DISTANCE

BoidScene::BoidScene(unsigned int numberOfBoids, Shader* shader, Mesh* mesh)
{
	InitGenerator(numberOfBoids);
#if CUDA
	hipSetDevice(0);
	BLOCKS_PER_GRID = (numberOfBoids + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK;

	m_Position = (glm::vec3*)malloc(numberOfBoids * sizeof(glm::vec3));
	m_Velocity = (glm::vec3*)malloc(numberOfBoids * sizeof(glm::vec3));

	hipHostMalloc((void**)&modelMatricesHostPinned, numberOfBoids * sizeof(glm::mat4));	
#endif

	glm::vec3 pos, vel;
	for (unsigned int i = 0; i < numberOfBoids; ++i)
	{
		pos = glm::vec3(rndX(), rndY(), rndZ());
		vel = glm::vec3(rndX(), rndY(), rndZ());

		Boid* b = new Boid(pos, vel);
		b->SetRenderComponent(new RenderComponent(mesh, shader));

#if CUDA
		m_Position[i] = pos;
		m_Velocity[i] = vel;
		b->SetWorldTransform(&modelMatricesHostPinned[i]);
#endif
		boids.push_back(b);
	}
	m_FlockHeading = glm::vec3(0, 0, 0);

#if CUDA
	hipMalloc((void**)&boidsDevice, sizeof(BoidGPU));
	hipMalloc((void**)&modelMatricesDevice, numberOfBoids * sizeof(glm::mat4));
	hipMemcpyAsync(boidsDevice->m_Position, m_Position, numberOfBoids * sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipMemcpyAsync(boidsDevice->m_Velocity, m_Velocity, numberOfBoids * sizeof(glm::vec3), hipMemcpyHostToDevice);

	//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
#endif

#if THREADED
	futures.clear();
#endif
}

BoidScene::~BoidScene()
{
#if CUDA
	hipDeviceSynchronize();
	hipFree(boidsDevice);
	hipFree(modelMatricesDevice);
	hipDeviceReset();
	free(m_Position);
	free(m_Velocity);
#endif
	for (auto& b : boids)
	{
		delete b;
		b = nullptr;
	}
	boids.clear();
}

void BoidScene::InitGenerator(int spread)
{
	std::random_device rD0;
	std::random_device rD1;
	std::random_device rD2;
	std::default_random_engine engine0(rD0());
	std::default_random_engine engine1(rD1());
	std::default_random_engine engine2(rD2());

	std::uniform_real_distribution<float> x(-100.0f, 100.0f);
	std::uniform_real_distribution<float> y(-100.0f, 100.0f);
	std::uniform_real_distribution<float> z(-100.0f, 100.0f);

	rndX = std::bind(x, engine0);
	rndY = std::bind(y, engine1);
	rndZ = std::bind(z, engine2);
}

void BoidScene::RenderScene()
{
	Scene::RenderScene();
	for (unsigned int i = 0; i < boids.size(); ++i)
		boids[i]->OnRenderObject();
}

#if !CUDA
#if THREADED
void BoidScene::UpdateScene(float dt)
{
	count += dt;
	if (count > 2500.0f)
	{
		m_FlockHeading = glm::vec3(rndX(), rndY(), rndZ());
		Boid::UpdateFlockHeading(m_FlockHeading);
		count = 0.0f;
	}

	size_t distribution = boids.size() / NUMBER_OF_THREADS;
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 0, distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, distribution + 1, 2 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 2 * distribution + 1, 3 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 3 * distribution + 1, 4 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 4 * distribution + 1, 5 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 5 * distribution + 1, 6 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 6 * distribution + 1, 7 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 7 * distribution + 1, boids.size() - 1, dt));

	for (auto& future : futures)
	{
		future.get();
	}
	futures.clear();

	Scene::UpdateScene(dt);
}

void BoidScene::UpdatePartition(size_t begin, size_t end, float dt)
{
	glm::vec3 posA, posB;
	for (unsigned int i = begin; i <= end; ++i)
	{
		posA = boids[i]->GetPosition();
		for (unsigned int j = 0; j < boids.size(); ++j)
		{
			if (i != j)
			{
				posB = boids[j]->GetPosition();
				float dist = glm::distance(posA, posB);
				if (dist <= MAX_DISTANCE)
				{
					BoidNeighbour bNA;
					bNA.n = boids[j];
					bNA.dist = dist;
					boids[i]->AddNeighbour(bNA);
				}
			}
		}
		boids[i]->OnUpdateObject(dt);
	}
}
#else
void BoidScene::UpdateScene(float dt)
{
	count += dt;
	if (count > 2000.0f)
	{
		m_FlockHeading = glm::vec3(rndX(), rndY(), rndZ());
		count = 0.0f;
	}

	glm::vec3 posA, posB;
	for (unsigned int i = 0; i < boids.size() - 1; ++i)
	{
		posA = boids[i]->GetPosition();
		for (unsigned int j = i + 1; j < boids.size(); ++j)
		{
			if (i != j)
			{
				posB = boids[j]->GetPosition();
				float dist = glm::length(posA - posB);
				BoidNeighbour bNA;
				bNA.n = boids[j];
				bNA.dist = dist;
				boids[i]->AddNeighbour(bNA);

				BoidNeighbour bNB;
				bNB.n = boids[i];
				bNB.dist = dist;
				boids[j]->AddNeighbour(bNB);
			}
		}
	}

	for (unsigned int i = 0; i < boids.size(); ++i)
		boids[i]->OnUpdateObject(dt);

	Scene::UpdateScene(dt);
}
#endif
#else
void BoidScene::UpdateScene(float dt)
{
	ComputeKNN << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(boidsDevice);
	CalcVelocity << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(boidsDevice, m_FlockHeading);
	UpdateBoid << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(boidsDevice, modelMatricesDevice, dt);
	hipMemcpyAsync(modelMatricesHostPinned, modelMatricesDevice, sizeof(glm::mat4) * NUM_BOIDS, hipMemcpyDeviceToHost);
	
	count += dt;
	if (count > 2500.0f)
	{
		m_FlockHeading = glm::vec3(rndX(), rndY(), rndZ());
		count = 0.0f;
	}

	Scene::UpdateScene(dt);
}

/*__global__ void ComputeKNN(BoidGPU* boid)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= NUM_BOIDS)
		return;

	glm::vec3 myPos = boid->m_Position[tid];
	glm::vec3 myVel = boid->m_Velocity[tid];
	glm::vec3 temp_cohVec = glm::vec3(0, 0, 0);
	glm::vec3 temp_sepVec = glm::vec3(0, 0, 0);
	glm::vec3 temp_alignVec = glm::vec3(0, 0, 0);	
	glm::vec3 dir = glm::vec3(0,0,0);
	float counter = 0;
	float dist = 0.0f;

#pragma unroll	
	for (unsigned int i = 0; i < NUM_BOIDS; ++i)
	{
		if (tid == i) continue;

		dir = boid->m_Position[i] - myPos;
		dist = glm::dot(dir, dir);

		if (dist <= MAX_DISTANCE_SQR)
		{
			counter += 1.0f;
			temp_cohVec += boid->m_Position[i];
			temp_sepVec -= dir * (1.0f / sqrtf(dist));
			temp_alignVec += boid->m_Velocity[i];
		}
	}	

	boid->m_CohesiveVector[tid] = (((temp_cohVec / counter) - myPos) * 0.0001f) - myVel;
	boid->m_SeperationVector[tid] = (temp_sepVec / counter) * 0.25f;
	boid->m_AlignmentVector[tid] = ((temp_alignVec / counter) - myVel);
}*/

__global__ void ComputeKNN(BoidGPU* boid)
{
	__shared__ glm::vec3 shPos[THREADS_PER_BLOCK];
	__shared__ glm::vec3 shVel[THREADS_PER_BLOCK];
	
	int gid = threadIdx.x + (blockIdx.x * blockDim.x);

	//if (tid < NUM_BOIDS)
	{
		int idx = threadIdx.x;
		float counter = 0.0f;
		float dist = 0.0f;
		glm::vec3 dir(0, 0, 0);
		glm::vec3 myPos = boid->m_Position[gid];
		glm::vec3 myVel = boid->m_Velocity[gid];
		glm::vec3 temp_cohVec(0, 0, 0);
		glm::vec3 temp_sepVec(0, 0, 0);
		glm::vec3 temp_alignVec(0, 0, 0);

#pragma unroll
		for (int i = 0, tile = 0; i < NUM_BOIDS; i += THREADS_PER_BLOCK, tile++)
		{
			idx = tile * blockDim.x + threadIdx.x;
			shPos[threadIdx.x] = boid->m_Position[idx];
			shVel[threadIdx.x] = boid->m_Velocity[idx];
			//idx += THREADS_PER_BLOCK;
			__syncthreads();

#pragma unroll
			for (int j = 0; j < THREADS_PER_BLOCK; ++j)
			{
				dir = shPos[j] - myPos;
				dist = glm::dot(dir, dir);
				if (dist <= MAX_DISTANCE_SQR)
				{
					if (dist < 0.00001f) continue;

					counter += 1.0f;
					temp_cohVec += shPos[j];	
					temp_sepVec -= dir * (1.0f / sqrtf(dist));
					temp_alignVec += shVel[j];
				}
			}
			__syncthreads();
		}
		
		boid->m_CohesiveVector[gid] = (((temp_cohVec / counter) - myPos) * 0.0001f) - myVel;
		boid->m_SeperationVector[gid] = (temp_sepVec / counter) * 0.25f;
		boid->m_AlignmentVector[gid] = ((temp_alignVec / counter) - myVel);
	}
}

__global__ void CalcVelocity(BoidGPU* boid, const glm::vec3 heading)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= NUM_BOIDS)
		return;

	glm::vec3 velocity = boid->m_CohesiveVector[tid] + boid->m_SeperationVector[tid] + boid->m_AlignmentVector[tid] + ((heading - boid->m_Position[tid]) * 0.001f);
	float speed = sqrtf(glm::dot(velocity, velocity));
	
	if (speed > 0.3f)
	{
		velocity = (velocity / speed) * 0.3f;
	}

	boid->m_Velocity[tid] = velocity * 0.999f;
	velocity = glm::normalize(velocity);
	boid->m_Rotation[tid] = glm::quat(glm::vec3(velocity.x, velocity.y, velocity.z));
}

__global__ void UpdateBoid(BoidGPU* boid, glm::mat4* boidMat, const float dt)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= NUM_BOIDS)
		return; 	

	boid->m_Position[tid] += (boid->m_Velocity[tid] * dt);
	boidMat[tid] = glm::mat4_cast(boid->m_Rotation[tid]) * glm::translate(boid->m_Position[tid]);
}
#endif