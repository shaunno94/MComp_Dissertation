#include "hip/hip_runtime.h"
#include "BoidScene.h"
#include "Common.h"
#include <iostream>

#if CUDA
#define MAX_NEIGHBOURS 150
#define THREADS_PER_BLOCK 256
const uint32_t THREADS_K1 = 16;
const uint32_t BLOCKS_K1 = (NUM_BOIDS + (THREADS_K1 - 1)) / THREADS_K1;
const dim3 THREAD_DIM_K1 = dim3(THREADS_K1, THREADS_K1);
const dim3 BLOCK_DIM_K1 = dim3(BLOCKS_K1, BLOCKS_K1);
#endif
//k value
#define MAX_DISTANCE 65.0f
#define MAX_DISTANCE_SQR MAX_DISTANCE * MAX_DISTANCE
BoidScene::BoidScene(unsigned int numberOfBoids, Shader* shader, Mesh* mesh)
{
	InitGenerator(numberOfBoids);
#if CUDA
	hipSetDevice(0);
	BLOCKS_PER_GRID = (numberOfBoids + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK;
	boidsHost = (BoidGPU*)malloc(numberOfBoids * sizeof(BoidGPU));
	hipHostMalloc((void**)&modelMatricesHostPinned, numberOfBoids * sizeof(glm::mat4));
#endif

	glm::vec3 pos, vel;
	for (unsigned int i = 0; i < numberOfBoids; ++i)
	{
		pos = glm::vec3(rndX(), rndY(), rndZ());
		vel = glm::vec3(rndX(), rndY(), rndZ());

		Boid* b = new Boid(pos, vel);
		b->SetRenderComponent(new RenderComponent(mesh, shader));

#if CUDA
		boidsHost[i] = BoidGPU(MAX_NEIGHBOURS, pos, vel);
		b->SetWorldTransform(&modelMatricesHostPinned[i]);
#endif
		boids.push_back(b);
	}
	m_FlockHeading = glm::vec3(0, 0, 0);

#if CUDA
	hipMalloc((void**)&boidsDevice, numberOfBoids * sizeof(BoidGPU));
	hipMalloc((void**)&modelMatricesDevice, numberOfBoids * sizeof(glm::mat4));
	hipMemcpy(boidsDevice, boidsHost, numberOfBoids * sizeof(BoidGPU), hipMemcpyHostToDevice);
	//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
#endif

#if THREADED
	futures.clear();
#endif
}

BoidScene::~BoidScene()
{
#if CUDA
	hipDeviceSynchronize();
	hipFree(boidsDevice);
	hipFree(modelMatricesDevice);
	//hipFree(modelMatrices_hostPinned);
	hipDeviceReset();
	free(boidsHost);
#endif
	for (auto& b : boids)
	{
		delete b;
		b = nullptr;
	}
	boids.clear();
}

void BoidScene::InitGenerator(int spread)
{
	std::random_device rD0;
	std::random_device rD1;
	std::random_device rD2;
	std::default_random_engine engine0(rD0());
	std::default_random_engine engine1(rD1());
	std::default_random_engine engine2(rD2());

	std::uniform_real_distribution<float> x(-100.0f, 100.0f);
	std::uniform_real_distribution<float> y(-100.0f, 100.0f);
	std::uniform_real_distribution<float> z(-100.0f, 100.0f);

	rndX = std::bind(x, engine0);
	rndY = std::bind(y, engine1);
	rndZ = std::bind(z, engine2);
}

void BoidScene::RenderScene()
{
	Scene::RenderScene();
	for (unsigned int i = 0; i < boids.size(); ++i)
		boids[i]->OnRenderObject();
}

#if !CUDA
#if THREADED
void BoidScene::UpdateScene(float dt)
{
	count += dt;
	if (count > 2500.0f)
	{
		m_FlockHeading = glm::vec3(rndX(), rndY(), rndZ());
		Boid::UpdateFlockHeading(m_FlockHeading);
		count = 0.0f;
	}

	size_t distribution = boids.size() / NUMBER_OF_THREADS;
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 0, distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, distribution + 1, 2 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 2 * distribution + 1, 3 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 3 * distribution + 1, 4 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 4 * distribution + 1, 5 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 5 * distribution + 1, 6 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 6 * distribution + 1, 7 * distribution, dt));
	futures.push_back(std::async(std::launch::async, &BoidScene::UpdatePartition, this, 7 * distribution + 1, boids.size() - 1, dt));

	for (auto& future : futures)
	{
		future.get();
	}
	futures.clear();

	Scene::UpdateScene(dt);
}

void BoidScene::UpdatePartition(size_t begin, size_t end, float dt)
{
	glm::vec3 posA, posB;
	for (unsigned int i = begin; i <= end; ++i)
	{
		posA = boids[i]->GetPosition();
		for (unsigned int j = 0; j < boids.size(); ++j)
		{
			if (i != j)
			{
				posB = boids[j]->GetPosition();
				float dist = glm::distance(posA, posB);
				if (dist <= MAX_DISTANCE)
				{
					BoidNeighbour bNA;
					bNA.n = boids[j];
					bNA.dist = dist;
					boids[i]->AddNeighbour(bNA);
				}
			}
		}
		boids[i]->OnUpdateObject(dt);
	}
}
#else
void BoidScene::UpdateScene(float dt)
{
	count += dt;
	if (count > 2000.0f)
	{
		m_FlockHeading = glm::vec3(rndX(), rndY(), rndZ());
		count = 0.0f;
	}

	glm::vec3 posA, posB;
	for (unsigned int i = 0; i < boids.size() - 1; ++i)
	{
		posA = boids[i]->GetPosition();
		for (unsigned int j = i + 1; j < boids.size(); ++j)
		{
			if (i != j)
			{
				posB = boids[j]->GetPosition();
				float dist = glm::length(posA - posB);
				BoidNeighbour bNA;
				bNA.n = boids[j];
				bNA.dist = dist;
				boids[i]->AddNeighbour(bNA);

				BoidNeighbour bNB;
				bNB.n = boids[i];
				bNB.dist = dist;
				boids[j]->AddNeighbour(bNB);
			}
		}
	}

	for (unsigned int i = 0; i < boids.size(); ++i)
		boids[i]->OnUpdateObject(dt);

	Scene::UpdateScene(dt);
}
#endif
#else
void BoidScene::UpdateScene(float dt)
{
	//ComputeKNN << <BLOCK_DIM_K1, THREAD_DIM_K1 >> >(boidsDevice);
	ComputeKNN << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(boidsDevice);
	CalcCohesion << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(boidsDevice);
	CalcAlignment << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(boidsDevice);
	CalcSeperation << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(boidsDevice);
	CalcVelocity << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(boidsDevice, m_FlockHeading);
	UpdateBoid << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(boidsDevice, modelMatricesDevice, dt);
	hipMemcpyAsync(modelMatricesHostPinned, modelMatricesDevice, sizeof(glm::mat4) * NUM_BOIDS, hipMemcpyDeviceToHost);
	
	count += dt;
	if (count > 2500.0f)
	{
		m_FlockHeading = glm::vec3(rndX(), rndY(), rndZ());
		count = 0.0f;
	}

	Scene::UpdateScene(dt);
}

__global__ void ComputeKNN(BoidGPU* boid)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= NUM_BOIDS)
		return;

	glm::vec3 temp_pos = boid[tid].m_Position;
	Neighbour* temp_neighbour = boid[tid].neighbours;
	unsigned int counter = 0;
	float dist = 0.0f;
	glm::vec3 dir = glm::vec3(0,0,0);

#pragma unroll	
	for (unsigned int i = 0; i < NUM_BOIDS; ++i)
	{
		if (tid == i) continue;

		dir = boid[i].m_Position - temp_pos;
		dist = glm::dot(dir, dir);
		if (dist <= MAX_DISTANCE_SQR)
		{
			temp_neighbour[counter].n = i;
			temp_neighbour[counter++].distance = 1.0f / sqrtf(dist);
			if (counter >= MAX_NEIGHBOURS) break;
		}
	}	

	boid[tid].neighbours = temp_neighbour;
	boid[tid].lastIndex = counter;
}

/*__global__ void ComputeKNN(BoidGPU* boid)
{
	__shared__ glm::vec3 temp_pos[THREADS_K1];
	//__shared__ unsigned int temp_index[THREADS_K1];
	int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
	int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
	temp_pos[threadIdx.x] = boid[tid_x].m_Position;
	//temp_index[threadIdx.x] = boid[tid_x].lastIndex;
	__syncthreads();
	
	if (tid_x < NUM_BOIDS && tid_y < NUM_BOIDS && tid_x != tid_y)
	{
		//if (glm::distance(boid[tid_x].m_Position, boid[tid_y].m_Position) <= MAX_DISTANCE)
		float dist = glm::distance(temp_pos[threadIdx.x], temp_pos[threadIdx.y]);
		if (dist <= MAX_DISTANCE)
		{
			unsigned int index = atomicAdd(&boid[tid_x].lastIndex, 1);
			boid[tid_x].neighbours[index].n = tid_y;
			//boid[tid_x].neighbours[atomicAdd(&temp_index[threadIdx.x], 1)].n = tid_y;
			//boid[tid_x].neighbours[index].distance = 1.0f / dist;
		}
	}

	//__syncthreads();
	//if (tid_x < NUM_BOIDS && tid_y < NUM_BOIDS && tid_x != tid_y)
	{
		//boid[tid_x].lastIndex += temp_index[threadIdx.x];
		//atomicAdd(&boid[tid_x].lastIndex, temp_index[threadIdx.x]);
	}
}*/

__global__ void CalcCohesion(BoidGPU* boid)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= NUM_BOIDS)
		return;

	glm::vec3 temp_cohVec = glm::vec3(0, 0, 0);
	unsigned int index = boid[tid].lastIndex;

	for (unsigned int i = 0; i < index; ++i)
	{
		temp_cohVec += boid[boid[tid].neighbours[i].n].m_Position;
	}

	temp_cohVec = glm::normalize((temp_cohVec / float(index - 1)) - boid[tid].m_Position);
	temp_cohVec *= (0.3f * (glm::length(temp_cohVec) * 0.001f));
	boid[tid].m_CohesiveVector = (temp_cohVec - boid[tid].m_Velocity);
}

__global__ void CalcSeperation(BoidGPU* boid)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= NUM_BOIDS)
		return;

	glm::vec3 temp_pos = boid[tid].m_Position;
	glm::vec3 temp_sepVec = glm::vec3(0, 0, 0);
	unsigned int index = boid[tid].lastIndex;
	//glm::vec3 dir;

	for (unsigned int i = 0; i < index; ++i)
	{
		//dir = boid[boid[tid].neighbours[i].n].m_Position - temp_pos;
		temp_sepVec -= ((boid[boid[tid].neighbours[i].n].m_Position - temp_pos) * boid[tid].neighbours[i].distance);
		//temp_sepVec -= dir / sqrt(glm::dot(dir, dir));
	}

	boid[tid].m_SeperationVector = (temp_sepVec / float(index - 1)) * 0.25f;
}

__global__ void CalcAlignment(BoidGPU* boid)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= NUM_BOIDS)
		return;

	unsigned int index = boid[tid].lastIndex;
	glm::vec3 temp_alignVec = glm::vec3(0, 0, 0);

	for (unsigned int i = 0; i < index; ++i)
	{
		temp_alignVec += boid[boid[tid].neighbours[i].n].m_Velocity;
	}
	
	boid[tid].m_AlignmentVector = ((temp_alignVec / float(index - 1)) - boid[tid].m_Velocity);
}

__global__ void CalcVelocity(BoidGPU* boid, const glm::vec3 heading)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= NUM_BOIDS)
		return;

	glm::vec3 velocity = boid[tid].m_CohesiveVector + boid[tid].m_SeperationVector + boid[tid].m_AlignmentVector + ((heading - boid[tid].m_Position) * 0.001f);
	float speed = sqrtf(glm::dot(velocity, velocity));
	
	if (speed > 0.3f)
	{
		velocity = (velocity / speed) * 0.3f;
	}
	boid[tid].m_Velocity = velocity * 0.999f;
	velocity = glm::normalize(velocity);
	boid[tid].m_Rotation = glm::quat(glm::vec3(velocity.x, velocity.y, velocity.z));
}

__global__ void UpdateBoid(BoidGPU* boid, glm::mat4* boidMat, const float dt)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid >= NUM_BOIDS)
		return; 	

	boid[tid].m_OldPosition = boid[tid].m_Position;
	boid[tid].m_Position += (boid[tid].m_Velocity * dt);	
	boidMat[tid] = glm::mat4_cast(boid[tid].m_Rotation) * glm::translate(boid[tid].m_OldPosition + boid[tid].m_Position);
	boid[tid].lastIndex = 0;
}
#endif